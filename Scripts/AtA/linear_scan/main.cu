#include "hip/hip_runtime.h"

// c++ inclusions
#include <stdlib.h>
#include <stdio.h>
#include <random>
#include <string.h>
#include <fstream>
#include <math.h>
#include <iostream>
#include <string>
#include <queue>
#include <iomanip>
#include <set>
#include <algorithm> 
#include <thread>
#include <cstdint>
#include <utility>
#include <vector>

// thrust inclusions
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/sort.h>
#include <thrust/device_ptr.h>
#include <thrust/system/cuda/execution_policy.h> //for streams for thrust (added with Thrust v1.8)
#include <thrust/binary_search.h>
#include <thrust/execution_policy.h>

// custom inclusions
#include "omp.h"
#include "ls_gpu.h"
#include "ls_cpu.h"
#include "ls_utility.h"

int main( int argc, char **argv )
{
    omp_set_num_threads( NTHREADS );
    omp_set_nested(1);

    args commandline_args;

    int args_success = parse_args( &commandline_args, argc, &argv );

    if( !args_success )
        {
            report_args_failure();

            return EXIT_FAILURE;
        }

    uint64_t total_num_batches = commandline_args.N / commandline_args.batch_size;
    uint64_t num_cpu_batches   = total_num_batches * commandline_args.cpu_frac;
    uint64_t num_gpu_batches   = total_num_batches - num_cpu_batches;

    time_data data_creation;
    time_data cpu_only;
    time_data gpu_only;
    time_data total_time;

    std::vector<uint64_t> batch_indices;
    batch_indices.reserve( total_num_batches );

	////////////////
	//Turn on gpu
	printf("\nTurning on the GPU...\n");
	warm_up_gpu( 0 );

    uint64_t *data = (uint64_t*) malloc( sizeof( uint64_t ) * commandline_args.N );

    // report data
	printf( "\nSeed for random number generator: %d", commandline_args.seed );
	printf( "\nInput size: %lu", commandline_args.N );
	printf( "\nBatch size: %lu", commandline_args.batch_size );
    printf( "\nTotal number of batches: %lu\n", total_num_batches );
    printf( "\nFraction of batches sent to the CPU: %.2f\n", commandline_args.cpu_frac );
    printf( "Number of CPU Batches: %lu\n", num_cpu_batches );
    printf( "Number of GPU Batches: %lu\n", num_gpu_batches );

    assert( num_cpu_batches + num_gpu_batches == total_num_batches );

    data_creation.start = omp_get_wtime();
    generate_dataset( data, commandline_args.N, commandline_args.seed );
    data_creation.end = omp_get_wtime();

    printf( "Time to create dataset: %f\n", get_elapsed( &data_creation ) );

    cpu_only.start   = omp_get_wtime();
    gpu_only.start   = omp_get_wtime();
    total_time.start = omp_get_wtime();

    // 0'th item is maximum for CPU, each consecutive is for the maximum for each stream
    uint64_t maximums[ STREAMSPERGPU + 1 ] = { 0 };
    uint64_t global_max = 0;

    uint64_t *device_data     = nullptr;
    uint64_t *pinned_host     = nullptr;
    #pragma omp parallel sections
    {

        // cpu section
        #pragma omp section
        {
            uint64_t cpu_index = 0;
            uint64_t my_max    = 0;

            if( num_cpu_batches > 0 )
                {
                    #pragma omp parallel for private( cpu_index ) reduction( max:my_max )
                    for( cpu_index = 0; cpu_index < commandline_args.batch_size * num_cpu_batches; ++cpu_index )
                        {
                            if( data[ cpu_index ] > my_max )
                                {
                                    my_max = data[ cpu_index ];
                                }
                        }
                    maximums[ 0 ] = my_max;

                    cpu_only.end = omp_get_wtime();
                }
        }

        // gpu section
        #pragma omp section
        {
            uint64_t gpu_index = 0;

            if( num_gpu_batches > 0 )
                {
                    hipError_t result = hipSuccess;
                    hipStream_t streams[ STREAMSPERGPU ];
                    uint64_t *device_maximums = nullptr;
                    uint64_t iterations_per_thread = num_gpu_batches / STREAMSPERGPU;

                    uint64_t batch_size = commandline_args.batch_size;
                    uint64_t transferred_so_far = 0;
                    const uint64_t total_blocks = ceil( batch_size * 1.0 / 1024.0 );
                    uint64_t *dev_batch_size = nullptr;

                    result = create_streams( streams, STREAMSPERGPU );
                    assert( result == hipSuccess );

                    // allocate enough STREAMSPERGPU batches + STREAMSPERGPU maximums, one max for each stream + one element to store batch_size
                    result = hipMalloc( &device_data, sizeof( uint64_t ) * ( ( batch_size * STREAMSPERGPU ) + STREAMSPERGPU + 1 ) );
                    assert( result == hipSuccess );

                    device_maximums = device_data + ( batch_size * STREAMSPERGPU );

                    result = hipHostMalloc( &pinned_host, sizeof( uint64_t ) * PINNEDBUFFER * STREAMSPERGPU );
                    assert( result == hipSuccess );

                    // copy the batch size to device
                    std::memcpy( pinned_host, &batch_size, sizeof( uint64_t ) );
                    result = hipMemcpyAsync( device_data + ( batch_size * STREAMSPERGPU ) + STREAMSPERGPU,
                                              pinned_host,
                                              sizeof( uint64_t ),
                                              hipMemcpyHostToDevice,
                                              streams[ 0 ]
                                              );

                    // synchronize and handle any errors 
                    hipStreamSynchronize( streams[ 0 ] );
                    assert( result == hipSuccess );                        

                    dev_batch_size = device_data + ( batch_size * STREAMSPERGPU ) + STREAMSPERGPU;
                               
      
                    #pragma omp parallel for num_threads( STREAMSPERGPU ) shared( pinned_host, device_data, streams, device_maximums ) \
                                             private( result, gpu_index, transferred_so_far )
                    for( gpu_index = num_cpu_batches; gpu_index < total_num_batches; ++gpu_index )
                        {
                            int thread_id = omp_get_thread_num();
                            int stream_id = thread_id;
                            uint64_t left_to_copy = batch_size;                    
                    
                            // device (start/end) pointers for a stream's batch
                            uint64_t *batch_start_ptr = device_data + ( stream_id * batch_size );                        
                            uint64_t *batch_end_ptr   = device_data + ( stream_id * batch_size ) + batch_size - 1;

                            // copy batch to pinned buffer in pinned_buffer_size chunks
                            // note: batch size may exceed size of pinned buffer, i.e., when N >= 3 x 10^9
                            
                            uint64_t size_to_transfer = 0;
                            transferred_so_far  = 0;

                            while( left_to_copy > 0 )
                            { 
                                // left_to_copy initially starting at batch_size
                                size_to_transfer = std::min( (uint64_t) PINNEDBUFFER, left_to_copy );

                                // copy to pinned buffer
                                std::memcpy( pinned_host + ( stream_id * PINNEDBUFFER ),
                                             data + ( gpu_index * batch_size ) + transferred_so_far,
                                             size_to_transfer * sizeof( uint64_t )
                                           ); 
                                
                                // copy to device
                                result = hipMemcpyAsync( device_data + ( stream_id * batch_size ) + transferred_so_far,
                                                          pinned_host + ( stream_id * PINNEDBUFFER ),
                                                          size_to_transfer * sizeof( uint64_t ),
                                                          hipMemcpyHostToDevice,
                                                          streams[ stream_id ]
                                                        );

                                // synchronize and handle any errors 
                                hipStreamSynchronize( streams[ stream_id ] );
                                assert( result == hipSuccess );                        
                               
                                left_to_copy -= size_to_transfer;
                                transferred_so_far += size_to_transfer;
                            }

                            // now, find the max element for my batch
                            kernel_max<<<total_blocks, 1024, 0, streams[ stream_id ]>>>( (unsigned long long int*) device_data + ( stream_id * batch_size ),
                                                                                         (unsigned long long int*) device_maximums + stream_id, (unsigned long long int*) dev_batch_size
                                                                                       );
                        }

                        // let stream 0 (default) transfer all maximums over        
                        result = hipMemcpyAsync( pinned_host,
                                                  device_maximums,
                                                  STREAMSPERGPU * sizeof( uint64_t ),
                                                  hipMemcpyDeviceToHost,
                                                  // stream 0 by default
                                                  streams[ 0 ]
                                                );

                        // synchronize and handle any errors
                        hipStreamSynchronize( 0 );
                        assert( result == hipSuccess );
                            
                        std::memcpy( maximums + 1, // first element is from the CPU
                                     pinned_host,
                                     STREAMSPERGPU * sizeof( uint64_t )
                                   );
                    }
            gpu_only.end = omp_get_wtime();
            }        

    }

    total_time.end = omp_get_wtime();

    uint64_t max_index = 0;
    for( max_index = 0; max_index < STREAMSPERGPU + 1; ++max_index )
        {
            if( maximums[ max_index ] > global_max )
                {
                    global_max = maximums[ max_index ];
                }
        }
        
    printf( "Max: %lu\n", global_max );
    printf( "CPU only time:  %f\n", get_elapsed( &cpu_only ) );
    printf( "GPU only time:  %f\n", get_elapsed( &gpu_only ) );
    printf( "Total time:     %f\n", get_elapsed( &total_time ) );
    printf( "Load imbalance: %f\n", abs( get_elapsed( &cpu_only ) - get_elapsed( &gpu_only ) ) / get_elapsed( &total_time ) );

    free( data );
    hipFree( device_data );
    hipHostFree( pinned_host );

    return EXIT_SUCCESS;
}
