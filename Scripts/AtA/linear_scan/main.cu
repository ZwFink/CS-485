#include "hip/hip_runtime.h"

// c++ inclusions
#include <stdlib.h>
#include <stdio.h>
#include <random>
#include <string.h>
#include <fstream>
#include <math.h>
#include <iostream>
#include <string>
#include <queue>
#include <iomanip>
#include <set>
#include <algorithm> 
#include <thread>
#include <cstdint>
#include <utility>
#include <vector>

// thrust inclusions
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/sort.h>
#include <thrust/device_ptr.h>
#include <thrust/system/cuda/execution_policy.h> //for streams for thrust (added with Thrust v1.8)
#include <thrust/binary_search.h>
#include <thrust/execution_policy.h>

// custom inclusions
#include "omp.h"
#include "ls_gpu.h"
#include "ls_cpu.h"
#include "ls_utility.h"

int main( int argc, char **argv )
{
    omp_set_num_threads( NTHREADS );
    omp_set_nested(1);

    args commandline_args;

    int args_success = parse_args( &commandline_args, argc, &argv );

    if( !args_success )
        {
            report_args_failure();

            return EXIT_FAILURE;
        }

    uint64_t total_num_batches = commandline_args.N / commandline_args.batch_size;
    uint64_t num_cpu_batches   = total_num_batches * commandline_args.cpu_frac;
    uint64_t num_gpu_batches   = total_num_batches - num_cpu_batches;

    time_data data_creation;
    time_data cpu_only;
    time_data gpu_only;
    time_data total_time;

    std::vector<uint64_t> batch_indices;
    batch_indices.reserve( total_num_batches );

	////////////////
	//Turn on gpu
	printf("\nTurning on the GPU...\n");
	warm_up_gpu( 0 );

    uint64_t *data = (uint64_t*) malloc( sizeof( uint64_t ) * commandline_args.N );

    // report data
	printf( "\nSeed for random number generator: %d", commandline_args.seed );
	printf( "\nInput size: %lu", commandline_args.N );
	printf( "\nBatch size: %lu", commandline_args.batch_size );
    printf( "\nTotal number of batches: %lu\n", total_num_batches );
    printf( "\nFraction of batches sent to the CPU: %.2f\n", commandline_args.cpu_frac );
    printf( "Number of CPU Batches: %lu\n", num_cpu_batches );
    printf( "Number of GPU Batches: %lu\n", num_gpu_batches );

    assert( num_cpu_batches + num_gpu_batches == total_num_batches );

    data_creation.start = omp_get_wtime();
    generate_dataset( data, commandline_args.N, commandline_args.seed );
    data_creation.end = omp_get_wtime();

    printf( "Time to create dataset: %f\n", get_elapsed( &data_creation ) );

    cpu_only.start   = omp_get_wtime();
    gpu_only.start   = omp_get_wtime();
    total_time.start = omp_get_wtime();

    compute_batches( &batch_indices, commandline_args.N, commandline_args.batch_size );

    // 0'th item is maximum for CPU, each consecutive is for the maximum for each stream
    uint64_t maximums[ STREAMSPERGPU + 1 ] = { 0 };
    uint64_t global_max = 0;

    uint64_t *device_data     = nullptr;
    uint64_t *pinned_host     = nullptr;
    #pragma omp parallel sections
    {

        // cpu section
        #pragma omp section
        {
            uint64_t cpu_index = 0;
            uint64_t my_max    = 0;

            if( num_cpu_batches > 0 )
                {
                    #pragma omp parallel for private( cpu_index ) reduction( max:my_max )
                    for( cpu_index = 0; cpu_index <= batch_indices[ num_cpu_batches - 1 ]; ++cpu_index )
                        {
                            if( data[ cpu_index ] > my_max )
                                {
                                    my_max = data[ cpu_index ];
                                }
                        }
                    maximums[ 0 ] = my_max;

                    cpu_only.end = omp_get_wtime();
                }
        }

        // gpu section
        #pragma omp section
        {
            uint64_t gpu_index = 0;

            if( num_gpu_batches > 0 )
                {
                    hipError_t result = hipSuccess;
                    hipStream_t streams[ STREAMSPERGPU ];
                    uint64_t *device_maximums = nullptr;
                    uint64_t iterations_per_thread = num_gpu_batches / STREAMSPERGPU;

                    uint64_t batch_size = commandline_args.batch_size;
                    uint64_t pinned_buffer_size = PINNEDBUFFER * STREAMSPERGPU;
                    uint64_t left_to_copy = batch_size;                    
                    uint64_t transferred_so_far = 0;

                    result = create_streams( streams, STREAMSPERGPU );
                    assert( result == hipSuccess );

                    // allocate enough STREAMSPERGPU batches + STREAMSPERGPU maximums, one max for each stream
                    result = hipMalloc( &device_data, sizeof( uint64_t ) * ( ( batch_size * STREAMSPERGPU ) + STREAMSPERGPU ) );
                    assert( result == hipSuccess );

                    // TODO: Check if this should be + 1
                    device_maximums = device_data + ( batch_size * STREAMSPERGPU );

                    result = hipHostMalloc( &pinned_host, sizeof( uint64_t ) * PINNEDBUFFER * STREAMSPERGPU );
                    assert( result == hipSuccess );
      
                    #pragma omp parallel for num_threads( STREAMSPERGPU ) shared( pinned_host, device_data, streams, device_maximums ) \
                                                                          private( result, gpu_index, index, left_to_copy )
                    for( gpu_index = num_cpu_batches; gpu_index < total_num_batches; ++gpu_index )
                        {
                            int thread_id = omp_get_thread_num();
                            int stream_id = thread_id % STREAMSPERGPU;
                    
                            // device (start/end) pointers for a stream's batch
                            uint64_t *batch_start_ptr = device_data + ( stream_id * batch_size );                        
                            uint64_t *batch_end_ptr   = device_data + ( stream_id * batch_size ) + batch_size - 1;

                            // copy batch to pinned buffer in pinned_buffer_size chunks
                            // note: batch size may exceed size of pinned buffer, i.e., when N >= 3 x 10^9
                            
                            // left_to_copy initially starting at batch_size
                            size_to_transfer = std::min( pinned_buffer_size, left_to_copy );

                            while( left_to_copy > 0 )
                            { 
                                // copy to pinned buffer
                                std::memcpy( pinned_host + ( stream_id * pinned_buffer_size ),
                                             data + ( gpu_index * batch_size ) + transferred_so_far,
                                             size_to_transfer * sizeof( uint64_t )
                                           ); 
                                
                                // copy to device
                                result = hipMemcpyAsync( device_data + ( stream_id * batch_size ) + transferred_so_far,
                                                          pinned_host + ( stream_id * pinned_buffer_size ),
                                                          size_to_transfer * sizeof( uint64_t ),
                                                          hipMemcpyHostToDevice,
                                                          streams[ stream_id ]
                                                        );

                                // synchronize and handle any errors 
                                hipStreamSynchronize( streams[ stream_id ] );
                                assert( result == hipSuccess );                        
                               
                                left_to_copy -= pinned_buffer_size;
                                transferred_so_far += size_to_transfer;
                                size_to_transfer = std::min( pinned_buffer_size, left_to_copy );
                            }

                            // now, find the max element for my batch
                            thrust::device_vector< uint64_t > dev_vector( batch_start_ptr, batch_end_ptr );
                            thrust::device_vector< uint64_t >::iterator iter = thrust::max_element( dev_vector.begin(), dev_vector.end() );
                            
                            // Do I have a larger max than my previous max stored?
                            if( *iter > *( device_maximums + stream_id ) )
                            {
                                *( device_maximums + stream_id ) = *iter;
                            }
                        }

                        // let stream 0 (default) transfer all maximums over        
                        result = hipMemcpyAsync( pinned_host + ( stream_id * pinned_buffer_size ),
                                                  device_maximums,
                                                  STREAMSPERGPU * sizeof( uint64_t ),
                                                  hipMemcpyDeviceToHost,
                                                  // stream 0 by default
                                                );

                        // synchronize and handle any errors
                        hipStreamSynchronize( streams[ stream_id ] );
                        assert( result == hipSuccess );
                            
                        std::memcpy( maximums + 1, // first element is from the CPU
                                     pinned_host + ( stream_id + pinned_buffer_size ),
                                     STREAMSPERGPU * sizeof( uint64_t )
                                   );
                    }
            }        

    }

    uint64_t max_index = 0;
    for( max_index = 0; max_index < STREAMSPERGPU + 1; ++max_index )
        {
            if( maximums[ max_index ] > global_max )
                {
                    global_max = maximums[ max_index ];
                }
        }
        
    printf( "Max: %lu\n", global_max );
    printf( "CPU only time: %f\n", get_elapsed( &cpu_only ) );

    free( data );
    hipFree( device_data );
    hipHostFree( pinned_host );

    return EXIT_SUCCESS;
}
