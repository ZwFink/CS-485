#include "hip/hip_runtime.h"
// c++ inclusions
#include <stdlib.h>
#include <stdio.h>
#include <random>
#include <string.h>
#include <fstream>
#include <math.h>
#include <iostream>
#include <string>
#include <queue>
#include <iomanip>
#include <set>
#include <algorithm> 
#include <thread>
#include <cstdint>
#include <utility>
#include <vector>

// thrust inclusions
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/sort.h>
#include <thrust/device_ptr.h>
#include <thrust/system/cuda/execution_policy.h> //for streams for thrust (added with Thrust v1.8)
#include <thrust/binary_search.h>
#include <thrust/execution_policy.h>

// custom inclusions
#include "omp.h"
#include "mm_cpu.h"
#include "mm_gpu.h"
#include "mm_utility.h"

int main( int argc, char **argv )
{
    uint64_t index, cpu_index, gpu_index;
    unsigned int numCPUBatches, numGPUBatches;
	
    omp_set_num_threads(NTHREADS);
	omp_set_nested(1);

	////////////////
	//Turn on gpu
	printf("\nTurning on the GPU...\n");
	warm_up_gpu( 0 );
	
	/////////////////////////
	// Get information from command line
	//1) the seed for random number generator
	/////////////////////////
	
	//Read in parameters from file:
	//dataset filename and cluster instance file
	if ( argc != 5 )
	{
		printf( "\n\nIncorrect number of input parameters.  \nShould include a seed for the random number generator, "
				"the number of elements, N, the batch size, and the number of lists, K\n"
		      );
		return 0;
	}
	
	//copy parameters from commandline:
	//char inputFname[]="data/test_data_removed_nan.txt";	
	
	char inputseed[ 500 ];
	strcpy( inputseed, argv[ 1 ] );

	char inputN[ 500 ];
	strcpy( inputN, argv[ 2 ] );

	char inputBatchSize[ 500 ];
	strcpy( inputBatchSize, argv[ 3 ] );
	
	unsigned int seed = atoi( inputseed );
	
	// uint64_t N=atoi(inputN);
	uint64_t N = strtoull( inputN, NULL, 0 );

	uint64_t BATCH_SIZE = strtoull( inputBatchSize, NULL, 0 );

	uint16_t K = strtoul( argv[ 4 ], NULL, 0 );

    uint64_t sublist_size = N / K;
	
    // helper vectors
    std::vector<uint64_t> first_sublist_starts;
    std::vector<uint64_t> first_sublist_ends;
    std::vector<uint64_t *> list_begin_ptrs; 
    std::vector<uint64_t> temp_start;
	std::vector<uint64_t> temp_end;
    // start and end vectors containing start and end
    // pivot vectors for each sublist
    std::vector<std::vector<uint64_t>> start_vectors;
    std::vector<std::vector<uint64_t>> end_vectors;
    
    // initialize array of integers
    uint64_t *input      = ( uint64_t * ) malloc( sizeof( uint64_t ) * N );
    uint64_t *output_arr = (uint64_t *) malloc( sizeof( uint64_t ) * N );

	printf( "\nSeed for random number generator: %d", seed );
	printf( "\nInput size: %lu", N );
	printf( "\nBatch size: %lu\n", BATCH_SIZE );
    printf( "K (number of sublists): %u\n", K );

    printf( "\nTotal size of input sorted array (MiB): %f", ((double) N * (sizeof(uint64_t)))/(1024.0*1024.0) );

    // Generate sorted sublists 
	double tstartsort = omp_get_wtime();
    list_begin_ptrs = *generate_k_sorted_sublists( input, N, seed, K );
	double tendsort = omp_get_wtime();

	printf( "\nTime to create K sorted sublists (not part of performance measurements): %f\n", tendsort - tstartsort );
	
	//start hybrid CPU + GPU total time timer
	double tstarthybrid = omp_get_wtime();
   
    // compute the number of batches
	compute_batches( sublist_size, input, &first_sublist_ends, BATCH_SIZE, sublist_size );
	
    // split the data between CPU and GPU for hybrid searches
	numCPUBatches = ( first_sublist_ends.size() - 1 ) * CPUFRAC;
	numGPUBatches = ( first_sublist_ends.size() - 1 ) - numCPUBatches;

    printf( "\nNumber of CPU batches: %u, Number of GPU batches: %u", numCPUBatches, numGPUBatches );
    assert( (numCPUBatches + numGPUBatches) == (first_sublist_ends.size() - 1) );

    // first_sublist_ends includes index 0 as first element which should be erased
    // note that this was only needed for calculation of num cpu/gpu batches.
    first_sublist_ends.erase( first_sublist_ends.begin() );
    
    // find start pivots for first sublist
    for( index = 0; index < sublist_size; index = index + BATCH_SIZE )
    {
        first_sublist_starts.push_back( index );
    }    
    
    start_vectors.push_back( first_sublist_starts );
    end_vectors.push_back( first_sublist_ends );

    
	// find remaining start and end pivot vectors for each sublist
	find_pivot_vectors( input, &start_vectors, &end_vectors, &first_sublist_ends, &list_begin_ptrs, sublist_size );

//	for( index = 0; index < end_vectors[1].size(); index++ )
//	{
//		printf( "\nbatch index: %lu\n", start_vectors[ 5 ][ index ] );
//	}
	
    #pragma omp parallel sections
    {
        
      // BEGIN CPU SECTION        
      #pragma omp section
      {

        for( cpu_index = 0; cpu_index < numCPUBatches; ++cpu_index )
        {
            // merge this round of batches
            multiwayMerge( &input, &output_arr, cpu_index, sublist_size, K, start_vectors, end_vectors );
        }

      }
            
      // BEGIN GPU SECTION
      #pragma omp section
      {
          hipStream_t streams[ STREAMSPERGPU ];
          const int NUM_THREADS_SEARCH = 4;
          hipError_t result = hipSuccess;
          std::vector<uint64_t> gpu_start_ptrs;
          std::vector<uint64_t> gpu_end_ptrs;
          uint64_t result_size = BATCH_SIZE * K * 2;
          uint64_t stream_size = BATCH_SIZE * K;
          uint64_t *output = nullptr;
          uint64_t *stream_dev_ptrs         = nullptr;
          uint64_t *input_to_gpu_pinned = nullptr;
          uint64_t *result_from_batches_pinned = nullptr;

          uint64_t gpu_output_index = get_gpu_output_index( &end_vectors, numCPUBatches, NUM_THREADS_SEARCH );

          gpu_start_ptrs.reserve( K );
          gpu_end_ptrs.reserve( K );

          result = create_streams( streams, STREAMSPERGPU );
          assert( result == hipSuccess );

          result = hipMalloc( (void**) &output, sizeof( uint64_t ) * result_size * 2 ); // 2 because we merge out of place
          assert( result == hipSuccess );

          result = hipMalloc( (void**) &stream_dev_ptrs, sizeof( uint64_t ) * stream_size );
          assert( result == hipSuccess );

          result = hipHostMalloc( (void**) &input_to_gpu_pinned, sizeof( uint64_t ) * BATCH_SIZE * STREAMSPERGPU );
          assert( result == hipSuccess );

          result = hipHostMalloc( (void**) &result_from_batches_pinned, sizeof( uint64_t ) * BATCH_SIZE * STREAMSPERGPU );
          assert( result == hipSuccess );

        for( gpu_index = numCPUBatches + 1; gpu_index <= numGPUBatches + numCPUBatches; ++gpu_index )
        {

            int thread_id = omp_get_thread_num();
            int stream_id = thread_id % STREAMSPERGPU;

            uint64_t start_index_gpu             = 0;
            uint64_t end_index_gpu               = 0;

            /*#pragma omp parallel for num_threads( STREAMSPERGPU ) schedule( static ) private( index, thread_id, stream_id, start_index_gpu, \
                        end_index_gpu, start_vectors, end_vectors ) \
                        shared ( K, gpu_index, numGPUBatches, numCPUBatches, result_from_batches_pinned, \
                                 input_to_gpu_pinned, stream_dev_ptrs, output, input \
                               ) */
            for( index = 0; index < K; index++ )
            {

                thread_id = omp_get_thread_num();
                stream_id = gpu_index % STREAMSPERGPU;

                start_index_gpu = 0;
                end_index_gpu   = 0;

                // copy data in BATCH_SIZE chunks from host memory to pinned memory
                start_index_gpu = start_vectors[ index ][ gpu_index ];
                end_index_gpu   = end_vectors[ index ][ gpu_index ];

                // calculate relative start
                gpu_start_ptrs[ index ] = gpu_index == numCPUBatches + 1 ? \
                                          0 : \
                                          start_vectors[ index ][ gpu_index ] - start_vectors[ index ][ gpu_index - 1 ];

                // calculate relative end index
                gpu_end_ptrs[ index ]   = gpu_index == numCPUBatches + 1 ? \
                                          start_vectors[ index ][ gpu_index ] - 1 : \
                                          end_vectors[ index ][ gpu_index ] - end_vectors[ index ][ gpu_index - 1 ];


                copy_to_device_buffer( input,
                                       input_to_gpu_pinned, stream_dev_ptrs,
                                       streams[ stream_id ],
                                       start_index_gpu, end_index_gpu,
                                       BATCH_SIZE, thread_id, stream_id
                                     );
                gpu_output_index += gpu_end_ptrs[ index ] - gpu_start_ptrs[ index ];
            }
                // do pairwise merging of sublists
            // merge the first two sublists, after the first merge we alternate
            // between output buffers
            thrust::merge( thrust::device, stream_dev_ptrs + gpu_start_ptrs[ 0 ],
                           stream_dev_ptrs + gpu_end_ptrs[ 0 ],
                           stream_dev_ptrs + gpu_start_ptrs[ 1 ],
                           stream_dev_ptrs + gpu_end_ptrs[ 1 ],
                           output
                         );

            #pragma omp parallel for num_threads( STREAMSPERGPU ) schedule( static ) private( index, thread_id, stream_id, start_index_gpu, \
                        end_index_gpu, start_vectors, end_vectors ) \
                        shared ( K, gpu_index, numGPUBatches, numCPUBatches, result_from_batches_pinned, \
                                 input_to_gpu_pinned, stream_dev_ptrs, output_arr, input, gpu_output_index, gpu_end_ptrs, gpu_start_ptrs \
                               )
            for( index = 0; index < K; index++ )
                {
                    thread_id = omp_get_thread_num();
                    stream_id = thread_id % STREAMSPERGPU;

                    // copy data in BATCH_SIZE chunks from device to host 
                    copy_from_device_buffer( output_arr + gpu_output_index,
                                             result_from_batches_pinned,
                                             output,
                                             streams[ stream_id ],
                                             BATCH_SIZE, thread_id, stream_id,
                                             &gpu_start_ptrs,
                                             &gpu_end_ptrs
                                             );
                }
        }

      }
    }

    // end hybrid CPU + GPU total time timer
	double tendhybrid = omp_get_wtime();
    free( input );
    free( output_arr );

	return EXIT_SUCCESS;

}
