#include "hip/hip_runtime.h"

// c++ inclusions
#include <stdlib.h>
#include <stdio.h>
#include <random>
#include <string.h>
#include <fstream>
#include <math.h>
#include <iostream>
#include <string>
#include <queue>
#include <iomanip>
#include <set>
#include <algorithm> 
#include <thread>
#include <cstdint>
#include <utility>
#include <vector>

// thrust inclusions
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/sort.h>
#include <thrust/device_ptr.h>
#include <thrust/system/cuda/execution_policy.h> //for streams for thrust (added with Thrust v1.8)
#include <thrust/binary_search.h>
#include <thrust/execution_policy.h>

// custom inclusions
#include "omp.h"
#include "mm_cpu.h"
#include "mm_gpu.h"
#include "mm_utility.h"

int main( int argc, char **argv )
{
    uint64_t index, cpu_index, gpu_index;
    unsigned int numCPUBatches, numGPUBatches;
	
    omp_set_num_threads(NTHREADS);
	omp_set_nested(1);

	////////////////
	//Turn on gpu
	printf("\nTurning on the GPU...\n");
	warm_up_gpu( 0 );
	
	/////////////////////////
	// Get information from command line
	//1) the seed for random number generator
	/////////////////////////
	
	//Read in parameters from file:
	//dataset filename and cluster instance file
	if ( argc != 5 )
	{
		printf( "\n\nIncorrect number of input parameters.  \nShould include a seed for the random number generator, "
				"the number of elements, N, the batch size, and the number of lists, K\n"
		      );
		return 0;
	}
	
	//copy parameters from commandline:
	char inputseed[ 500 ];
	strcpy( inputseed, argv[ 1 ] );

	char inputN[ 500 ];
	strcpy( inputN, argv[ 2 ] );

	char inputBatchSize[ 500 ];
	strcpy( inputBatchSize, argv[ 3 ] );
	
	unsigned int seed = atoi( inputseed );
	
	uint64_t N = strtoull( inputN, NULL, 0 );

	uint64_t BATCH_SIZE = strtoull( inputBatchSize, NULL, 0 );

	uint16_t K = strtoul( argv[ 4 ], NULL, 0 );

    uint64_t sublist_size = N / K;
	
    // helper vectors
    std::vector<uint64_t> first_sublist_starts;
    std::vector<uint64_t> first_sublist_ends;
    std::vector<uint64_t *> list_begin_ptrs; 
    // start and end vectors containing start and end
    // pivot vectors for each sublist
    std::vector<std::vector<uint64_t>> start_vectors;
    std::vector<std::vector<uint64_t>> end_vectors;
    
    // initialize array of integers
    uint64_t *input      = ( uint64_t * ) malloc( sizeof( uint64_t ) * N );
    uint64_t *output_arr = (uint64_t *) malloc( sizeof( uint64_t ) * N );

	printf( "\nSeed for random number generator: %d", seed );
	printf( "\nInput size: %lu", N );
	printf( "\nBatch size: %lu\n", BATCH_SIZE );
    printf( "K (number of sublists): %u\n", K );

    printf( "\nTotal size of input sorted array (MiB): %f", ((double) N * (sizeof(uint64_t)))/(1024.0*1024.0) );

    // Generate sorted sublists 
	double tstartsort = omp_get_wtime();
    list_begin_ptrs = *generate_k_sorted_sublists( input, N, seed, K );
	double tendsort = omp_get_wtime();

	printf( "\nTime to create K sorted sublists (not part of performance measurements): %f\n", tendsort - tstartsort );
	
	//start hybrid CPU + GPU total time timer
	double tstarthybrid = omp_get_wtime();

    double tstartgpu    = omp_get_wtime();
    double tendgpu      = 0;

    double tstartcpu    = omp_get_wtime();
    double tendcpu      = 0;
   
    // compute the number of batches
	compute_batches( sublist_size, input, &first_sublist_ends, BATCH_SIZE, sublist_size );
	
    // split the data between CPU and GPU for hybrid searches
	numCPUBatches = ( first_sublist_ends.size() - 1 ) * CPUFRAC;
	numGPUBatches = ( first_sublist_ends.size() - 1 ) - numCPUBatches;

    printf( "\nNumber of CPU batches: %u, Number of GPU batches: %u\n", numCPUBatches, numGPUBatches );
    assert( (numCPUBatches + numGPUBatches) == (first_sublist_ends.size() - 1) );

    // first_sublist_ends includes index 0 as first element which should be erased
    // note that this was only needed for calculation of num cpu/gpu batches.
    first_sublist_ends.erase( first_sublist_ends.begin() );
    
    // find start pivots for first sublist
    for( index = 0; index < sublist_size; index = index + BATCH_SIZE )
    {
        first_sublist_starts.push_back( index );
    }    
    
    start_vectors.push_back( first_sublist_starts );
    end_vectors.push_back( first_sublist_ends );

    
	// find remaining start and end pivot vectors for each sublist
	find_pivot_vectors( input, &start_vectors, &end_vectors, &first_sublist_ends, &list_begin_ptrs, sublist_size );

    #pragma omp parallel sections
    {
        
      // BEGIN CPU SECTION        
      #pragma omp section
      {
          // MULTIWAY MERGE BY LOOPING THROUGH SPLITS
         // #pragma omp parallel for num_threads( NTHREADS ) schedule( static ) private( cpu_index ) \
         //                        shared( input, output_arr, sublist_size, K, start_vectors, end_vectors )
         // for( cpu_index = 0; cpu_index < numCPUBatches; ++cpu_index )
         // {
         //     // merge this round of batches
         //     multiwayMergeBySplits( &input, &output_arr, cpu_index, sublist_size, K, start_vectors, end_vectors );
         // }

          // MULTIWAY MERGE ALL AT ONCE
          multiwayMerge( &input, &output_arr, numCPUBatches - 1, sublist_size, K, start_vectors, end_vectors );
          tendcpu = omp_get_wtime();
      }
            
      // BEGIN GPU SECTION
      #pragma omp section
      {
          hipStream_t streams[ STREAMSPERGPU ];
          const int NUM_THREADS_SEARCH = 4;
          hipError_t result = hipSuccess;
          std::vector<uint64_t> gpu_start_ptrs;
          std::vector<uint64_t> gpu_end_ptrs;
          uint64_t result_size = BATCH_SIZE * K * 2;
          uint64_t stream_size = BATCH_SIZE * K;
          uint64_t *output = nullptr;
          uint64_t *stream_dev_ptrs         = nullptr;
          uint64_t *input_to_gpu_pinned = nullptr;
          uint64_t *output_second = nullptr;
          uint64_t *result_from_batches_pinned = nullptr;

          tstartgpu = omp_get_wtime();

          uint64_t gpu_output_index = get_gpu_output_index( &end_vectors, numCPUBatches, NUM_THREADS_SEARCH );

          gpu_start_ptrs.reserve( K );
          gpu_end_ptrs.reserve( K );

          result = create_streams( streams, STREAMSPERGPU );
          assert( result == hipSuccess );

          result = hipMalloc( (void**) &output, sizeof( uint64_t ) * result_size * 2 ); // 2 because we merge out of place
          assert( result == hipSuccess );
          output_second = output + result_size;

          result = hipMalloc( (void**) &stream_dev_ptrs, sizeof( uint64_t ) * stream_size );
          assert( result == hipSuccess );

          result = hipHostMalloc( (void**) &input_to_gpu_pinned, sizeof( uint64_t ) * BATCH_SIZE * STREAMSPERGPU );
          assert( result == hipSuccess );

          result = hipHostMalloc( (void**) &result_from_batches_pinned, sizeof( uint64_t ) * BATCH_SIZE * STREAMSPERGPU );
          assert( result == hipSuccess );

          uint64_t *output_after_rounds = K % 2 ? output_second : output;

        for( gpu_index = numCPUBatches + 1; gpu_index <= numGPUBatches + numCPUBatches; ++gpu_index )
        {

            int thread_id = omp_get_thread_num();
            int stream_id = thread_id % STREAMSPERGPU;

            uint64_t start_index_gpu             = 0;
            uint64_t end_index_gpu               = 0;
            uint64_t merged_this_round = 0;

            #pragma omp parallel for num_threads( STREAMSPERGPU ) schedule( static ) private( index, thread_id, stream_id, start_index_gpu, \
                        end_index_gpu, start_vectors, end_vectors ) \
                        shared ( K, gpu_index, numGPUBatches, numCPUBatches, result_from_batches_pinned, \
                                 input_to_gpu_pinned, stream_dev_ptrs, output, input \
                               )
            for( index = 0; index < K; index++ )
            {

                thread_id = omp_get_thread_num();
                stream_id = gpu_index % STREAMSPERGPU;

                start_index_gpu = 0;
                end_index_gpu   = 0;

                // copy data in BATCH_SIZE chunks from host memory to pinned memory
                start_index_gpu = start_vectors[ index ][ gpu_index ];
                end_index_gpu   = end_vectors[ index ][ gpu_index ];

                // calculate relative start
                gpu_start_ptrs[ index ] = gpu_index == numCPUBatches + 1 ? \
                                          0 : \
                                          start_vectors[ index ][ gpu_index ] - start_vectors[ index ][ gpu_index - 1 ];

                // calculate relative end index
                gpu_end_ptrs[ index ]   = gpu_index == numCPUBatches + 1 ? \
                                          start_vectors[ index ][ gpu_index ] - 1 : \
                                          end_vectors[ index ][ gpu_index ] - end_vectors[ index ][ gpu_index - 1 ];


                copy_to_device_buffer( input,
                                       input_to_gpu_pinned, stream_dev_ptrs,
                                       streams[ stream_id ],
                                       start_index_gpu, end_index_gpu,
                                       BATCH_SIZE, thread_id, stream_id
                                     );
                gpu_output_index += gpu_end_ptrs[ index ] - gpu_start_ptrs[ index ];
            }
                // do pairwise merging of sublists
            // merge the first two sublists, after the first merge we alternate
            // between output buffers
            thrust::merge( thrust::device, stream_dev_ptrs + gpu_start_ptrs[ 0 ],
                           stream_dev_ptrs + gpu_end_ptrs[ 0 ],
                           stream_dev_ptrs + gpu_start_ptrs[ 1 ],
                           stream_dev_ptrs + gpu_end_ptrs[ 1 ],
                           output
                         );
            merged_this_round = gpu_end_ptrs[ 0 ] - gpu_start_ptrs[ 0 ] + \
                                gpu_end_ptrs[ 1 ] - gpu_start_ptrs[ 1 ];

            for( index = 2; index < K; ++index )
                {
                    if( index % 2 )
                        {
                            thrust::merge( thrust::device,
                                           output, output  + merged_this_round,
                                           stream_dev_ptrs + gpu_start_ptrs[ index ],
                                           stream_dev_ptrs + gpu_end_ptrs[ index ],
                                           output_second
                                         );
                        }
                    else
                        {
                            thrust::merge( thrust::device,
                                           output_second, output_second + merged_this_round,
                                           stream_dev_ptrs + gpu_start_ptrs[ index ],
                                           stream_dev_ptrs + gpu_end_ptrs[ index ],
                                           output
                                         );
                        }

                    merged_this_round += gpu_end_ptrs[ index ] - gpu_start_ptrs[ index ];
                }

            #pragma omp parallel for num_threads( STREAMSPERGPU ) schedule( static ) private( index, thread_id, stream_id, start_index_gpu, \
                        end_index_gpu, start_vectors, end_vectors ) \
                        shared ( K, gpu_index, numGPUBatches, numCPUBatches, result_from_batches_pinned, \
                                 input_to_gpu_pinned, stream_dev_ptrs, output_arr, input, gpu_output_index, gpu_end_ptrs, gpu_start_ptrs \
                               )
            for( index = 0; index < K; index++ )
                {
                    thread_id = omp_get_thread_num();
                    stream_id = thread_id % STREAMSPERGPU;

                    // copy data in BATCH_SIZE chunks from device to host 
                    copy_from_device_buffer( output_arr + gpu_output_index,
                                             result_from_batches_pinned,
                                             output_after_rounds,
                                             streams[ stream_id ],
                                             BATCH_SIZE, thread_id, stream_id,
                                             &gpu_start_ptrs,
                                             &gpu_end_ptrs
                                           );
                }
        }

        tendgpu = omp_get_wtime();
      }
    }

    // end hybrid CPU + GPU total time timer
	double tendhybrid = omp_get_wtime();

    double hybrid_total_time = tendhybrid - tstarthybrid;
    double cpu_total_time    = tendcpu    - tstartcpu;
    double gpu_total_time    = tendgpu    - tstartgpu;

    // formula given in paper
    double load_imbalance    = ( cpu_total_time - gpu_total_time ) / hybrid_total_time;
       
    printf( "Time CPU and GPU (total time): %f\n", hybrid_total_time );
    printf( "Time CPU Only: %f\n", cpu_total_time );
    printf( "Time GPU Only: %f\n", gpu_total_time );

    printf( "Load imbalance: %f\n", load_imbalance );

    free( input );
    free( output_arr );

	return EXIT_SUCCESS;

}
