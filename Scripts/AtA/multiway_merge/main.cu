#include "hip/hip_runtime.h"
// c++ inclusions
#include <stdlib.h>
#include <stdio.h>
#include <random>
#include <string.h>
#include <fstream>
#include <math.h>
#include <iostream>
#include <string>
#include <queue>
#include <iomanip>
#include <set>
#include <algorithm> 
#include <thread>
#include <cstdint>
#include <utility>
#include <vector>

// thrust inclusions
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/sort.h>
#include <thrust/device_ptr.h>
#include <thrust/system/cuda/execution_policy.h> //for streams for thrust (added with Thrust v1.8)
#include <thrust/binary_search.h>
#include <thrust/execution_policy.h>

// custom inclusions
#include "omp.h"
#include "mm_cpu.h"
#include "mm_gpu.h"
#include "mm_utility.h"

int main( int argc, char **argv )
{
    uint64_t index, cpu_index, gpu_index, curr_end_index = 0, piv_index, pivot_val;
    uint64_t *temp_ptr = nullptr;
    unsigned int numCPUBatches, numGPUBatches;
	
    omp_set_num_threads(NTHREADS);
	omp_set_nested(1);

	////////////////
	//Turn on gpu
	printf("\nTurning on the GPU...\n");
	warm_up_gpu( 0 );
	
	/////////////////////////
	// Get information from command line
	//1) the seed for random number generator
	/////////////////////////
	
	//Read in parameters from file:
	//dataset filename and cluster instance file
	if ( argc != 5 )
	{
		printf( "\n\nIncorrect number of input parameters.  \nShould include a seed for the random number generator, "
				"the number of elements, N, the batch size, and the number of lists, K\n"
		      );
		return 0;
	}
	
	//copy parameters from commandline:
	//char inputFname[]="data/test_data_removed_nan.txt";	
	
	char inputseed[ 500 ];
	strcpy( inputseed, argv[ 1 ] );

	char inputN[ 500 ];
	strcpy( inputN, argv[ 2 ] );

	char inputBatchSize[ 500 ];
	strcpy( inputBatchSize, argv[ 3 ] );
	
	unsigned int seed = atoi( inputseed );
	
	// uint64_t N=atoi(inputN);
	uint64_t N = strtoull( inputN, NULL, 0 );

	uint64_t BATCH_SIZE = strtoull( inputBatchSize, NULL, 0 );

	uint16_t K = strtoul( argv[ 4 ], NULL, 0 );

    uint64_t sublist_size = N / K;

	printf( "\nSeed for random number generator: %d", seed );
	printf( "\nInput size: %lu", N );
	printf( "\nBatch size: %lu\n", BATCH_SIZE );
    printf( "K (number of sublists): %u\n", K );

    // offset vectors  (NEEDS TO BE DELETED)
	//std::vector<uint64_t> first_sublist_pivots;
    //std::vector<uint64_t> offset_list_cpu;
    std::vector<uint64_t> offset_list_gpu;
    //std::vector<uint64_t> offset_begin_cpu;
    std::vector<uint64_t> offset_begin_gpu;
    
	// helper vectors
    std::vector<uint64_t> first_sublist_starts;
    std::vector<uint64_t> first_sublist_ends;
    std::vector<uint64_t *> list_begin_ptrs; 
    std::vector<uint64_t> temp_start;
	std::vector<uint64_t> temp_end;
    // start and end vectors containing start and end
    // pivot vectors for each sublist
    std::vector<std::vector<uint64_t>> start_vectors;
    std::vector<std::vector<uint64_t>> end_vectors;
    
    // initialize array of integers
    uint64_t *input      = ( uint64_t * ) malloc( sizeof( uint64_t ) * N );
    uint64_t *output_arr = (uint64_t *) malloc( sizeof( uint64_t ) * N );
    uint64_t *tempBuff   = ( uint64_t * ) malloc( sizeof( uint64_t ) * N );


    printf( "\nTotal size of input sorted array (MiB): %f", ((double) N * (sizeof(uint64_t)))/(1024.0*1024.0) );

    // Generate sorted sublists 
	double tstartsort = omp_get_wtime();
    list_begin_ptrs = *generate_k_sorted_sublists( input, N, seed, K );
	double tendsort = omp_get_wtime();

	printf( "\nTime to create K sorted sublists (not part of performance measurements): %f\n", tendsort - tstartsort );
	

	//start hybrid CPU + GPU total time timer
	double tstarthybrid = omp_get_wtime();
    
    // compute the number of batches
	// The number of batches should ensure that the input dataset is split at one point
	// The input batch size is thus an approximation
	compute_batches( sublist_size, input, &first_sublist_ends, BATCH_SIZE );
	
    // split the data between CPU and GPU for hybrid searches
	numCPUBatches = ( first_sublist_ends.size() - 1 ) * CPUFRAC;
	numGPUBatches = ( first_sublist_ends.size() - 1 ) - numCPUBatches;

    printf( "\nNumber of CPU batches: %u, Number of GPU batches: %u", numCPUBatches, numGPUBatches );
    assert( (numCPUBatches + numGPUBatches) == (first_sublist_ends.size() - 1) );

    // first_sublist_ends includes index 0 as first element which should be erased
    // note that this was only needed for calculation of num cpu/gpu batches.
    first_sublist_ends.erase( first_sublist_ends.begin() );
    
    // find start pivots for first sublist
	uint64_t iter = 0;
    for( index = 0; index < N; index = index + BATCH_SIZE )
    {
        first_sublist_starts[ iter ] = index;
		iter++;
    }    
    
    start_vectors[0] = first_sublist_starts;
    end_vectors[0] = first_sublist_ends;

    // find remaining start and end pivot vectors for each sublist
    // TO DO: create function find_pivot_vectors() for task below
    
	find_pivot_vectors( input, &start_vectors, &end_vectors, &first_sublist_ends, &list_begin_ptrs, sublist_size );

	//for( index = 0; index < list_begin_ptrs.size(); ++index )
    //{
    //    // create sublist pivot starts and pivot ends
    //    //temp_start = new std::vector<uint64_t>;
    //    //temp_end = new std::vector<uint64_t>;
	//	temp_start.clear();
	//	temp_end.clear();		


    //    for( piv_index = 0; piv_index < first_sublist_starts.size(); ++piv_index )
    //    {
    //        pivot_val = first_sublist_ends[ piv_index ];
    //
    //        temp_ptr = std::upper_bound( 
    //                                list_begin_ptrs + index, 
    //                                list_begin_ptrs + index + (sublist_size - 1), 
    //                                pivot_val 
    //                              );

    //        curr_end_index = thrust::distance( list_begin_ptrs, temp_ptr );

    //        temp_end->push_back( curr_end_index );

    //        if( piv_index == 0 )
    //        {
    //            temp_start->push_back( (*list_begin_ptrs)[ index ] );
    //        }

    //        else
    //        {
    //            temp_start->push_back( temp_end[ piv_index - 1 ] );
    //        }
    //    }

    //    start_vectors[ index ] = temp_start;
    //    end_vectors[ index ] = temp_end;
    //}





	

    #pragma omp parallel sections
    {
        
      // BEGIN CPU SECTION        
      #pragma omp section
      {
        //for( cpu_index = 1; cpu_index <= numCPUBatches; ++cpu_index )
        //{
        //    if( offset_list_cpu.size() == 0 )
        //    {
        //        set_beginning_of_offsets( &offset_begin_cpu, sublist_size, K );
        //    }

        //    else // copy over indices from offset_list to offset_begin
        //    {
        //        get_offset_beginning( &offset_list_cpu, &offset_begin_cpu );
        //        
        //        offset_list_cpu.clear();
        //    }

        //    // find offset_list_cpu 
        //    compute_offsets( input, &first_sublist_offsets, &offset_list_cpu, cpu_index, K, sublist_size ); 
    
        //    // merge this round of batches
        //    multiwayMerge( &input, &tempBuff, start_index, sublist_size, K, offset_begin_cpu, offset_list_cpu );
        //    
        //    // find start_index
        //    start_index = get_start_index( offset_list_cpu, K, sublist_size );

        //    // clear offset_list and offset_begin
        //    offset_begin_cpu.clear();
        //}

      }
            
      // BEGIN GPU SECTION
      #pragma omp section
      {
          hipStream_t streams[ STREAMSPERGPU ];
          hipError_t result = hipSuccess;
          uint64_t result_size = BATCH_SIZE * K * numGPUBatches;
          uint64_t stream_size = BATCH_SIZE * K;
          uint64_t *output = nullptr;
          uint64_t *stream_dev_ptrs         = nullptr;
          uint64_t *input_to_gpu_pinned = nullptr;
          uint64_t *result_from_batches_pinned = nullptr;

          result = create_streams( streams, STREAMSPERGPU );
          assert( result == hipSuccess );

          result = hipMalloc( (void**) &output, sizeof( uint64_t ) * result_size * 2 ); // 2 because we merge out of place
          assert( result == hipSuccess );

          result = hipMalloc( (void**) &stream_dev_ptrs, sizeof( uint64_t ) * stream_size );
          assert( result == hipSuccess );

          result = hipHostMalloc( (void**) &input_to_gpu_pinned, sizeof( uint64_t ) * BATCH_SIZE * STREAMSPERGPU );
          assert( result == hipSuccess );

          result = hipHostMalloc( (void**) &result_from_batches_pinned, sizeof( uint64_t * ) * BATCH_SIZE * STREAMSPERGPU );
          assert( result == hipSuccess );

        for( gpu_index = numCPUBatches + 1 ; gpu_index <= numGPUBatches + numCPUBatches; ++gpu_index )
        {

            int thread_id = omp_get_thread_num();
            int stream_id = thread_id % STREAMSPERGPU;

            uint64_t start_index_gpu = 0;
            uint64_t end_index_gpu   = 0;

            #pragma omp parallel for num_threads( STREAMSPERGPU ) schedule( static ) private( index, thread_id, stream_id, start_index_gpu, \
                        end_index_gpu, start_vectors, end_vectors ) \
                        shared ( K, gpu_index, numGPUBatches, numCPUBatches, result_from_batches_pinned, \
                                 input_to_gpu_pinned, stream_dev_ptrs, output, input \
                               )
            for( index = 0; index < K; index++ )
            {

                thread_id = omp_get_thread_num();
                stream_id = thread_id % STREAMSPERGPU;

                start_index_gpu = 0;
                end_index_gpu   = 0;

                // copy data in BATCH_SIZE chunks from host memory to pinned memory
                start_index_gpu = start_vectors[ index ][ gpu_index ];
                end_index_gpu   = end_vectors[ index ][ gpu_index ];

                copy_to_device_buffer( list_begin_ptrs[ index ],
                                       input_to_gpu_pinned, stream_dev_ptrs,
                                       streams[ stream_id ],
                                       start_index_gpu, end_index_gpu,
                                       BATCH_SIZE, thread_id, stream_id
                                     );
                // copy data in BATCH_SIZE chunks from pinned data to gpu
                // do pairwise merging of sublists

                // copy data in BATCH_SIZE chunks from device to host 
            }
        }
      }
    }

    // end hybrid CPU + GPU total time timer
	double tendhybrid = omp_get_wtime();
    free( input );
    free( output_arr );

	return EXIT_SUCCESS;

}
