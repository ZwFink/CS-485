
// c++ inclusions
#include <stdlib.h>
#include <stdio.h>
#include <random>
#include <string.h>
#include <fstream>
#include <math.h>
#include <iostream>
#include <string>
#include <queue>
#include <iomanip>
#include <set>
#include <algorithm> 
#include <thread>
#include <cstdint>
#include <utility>
#include <vector>

// thrust inclusions
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/sort.h>
#include <thrust/device_ptr.h>
#include <thrust/system/cuda/execution_policy.h> //for streams for thrust (added with Thrust v1.8)
#include <thrust/binary_search.h>
#include <thrust/execution_policy.h>

// custom inclusions
#include "omp.h"
#include "mm_cpu.h"
#include "mm_gpu.h"
#include "mm_utility.h"

int main( int argc, char **argv )
{
    uint64_t index, gpu_index;
    unsigned int numCPUBatches, numGPUBatches;
	
    omp_set_num_threads(NTHREADS);
	omp_set_nested(1);

	////////////////
	//Turn on gpu
	printf("\nTurning on the GPU...\n");
	warm_up_gpu( 0 );
	
	/////////////////////////
	// Get information from command line
	//1) the seed for random number generator
	/////////////////////////
	
	//Read in parameters from file:
	//dataset filename and cluster instance file
	if ( argc != 5 )
	{
		printf( "\n\nIncorrect number of input parameters.  \nShould include a seed for the random number generator, "
				"the number of elements, N, the batch size, and the number of lists, K\n"
		      );
		return 0;
	}
	
	//copy parameters from commandline:
	char inputseed[ 500 ];
	strcpy( inputseed, argv[ 1 ] );

	char inputN[ 500 ];
	strcpy( inputN, argv[ 2 ] );

	char inputBatchSize[ 500 ];
	strcpy( inputBatchSize, argv[ 3 ] );
	
	unsigned int seed = atoi( inputseed );
	
	uint64_t N = strtoull( inputN, NULL, 0 );

	uint64_t BATCH_SIZE = strtoull( inputBatchSize, NULL, 0 );

	uint16_t K = strtoul( argv[ 4 ], NULL, 0 );

    uint64_t sublist_size = N / K;
	
    // helper vectors
    std::vector<uint64_t> first_sublist_starts;
    std::vector<uint64_t> first_sublist_ends;
    std::vector<uint64_t *> list_begin_ptrs; 
    // start and end vectors containing start and end
    // pivot vectors for each sublist
    std::vector<std::vector<uint64_t>> start_vectors;
    std::vector<std::vector<uint64_t>> end_vectors;
    
    // initialize array of integers
    uint64_t *input      = ( uint64_t * ) malloc( sizeof( uint64_t ) * N );
    uint64_t *output_arr = (uint64_t *) malloc( sizeof( uint64_t ) * N );

	printf( "\nSeed for random number generator: %d", seed );
	printf( "\nInput size: %lu", N );
	printf( "\nBatch size: %lu\n", BATCH_SIZE );
    printf( "K (number of sublists): %u\n", K );

    printf( "\nTotal size of input sorted array (MiB): %f", ((double) N * (sizeof(uint64_t)))/(1024.0*1024.0) );

    // Generate sorted sublists 
	double tstartsort = omp_get_wtime();
    list_begin_ptrs = *generate_k_sorted_sublists( input, N, seed, K );
	double tendsort = omp_get_wtime();

	printf( "\nTime to create K sorted sublists (not part of performance measurements): %f\n", tendsort - tstartsort );
	
	//start hybrid CPU + GPU total time timer
	double tstarthybrid = omp_get_wtime();

    double tstartgpu    = omp_get_wtime();
    double tendgpu      = 0;

    double tstartcpu    = omp_get_wtime();
    double tendcpu      = 0;
   
    // compute the number of batches
	compute_batches( sublist_size, input, &first_sublist_ends, BATCH_SIZE, sublist_size );
	
    // split the data between CPU and GPU for hybrid searches
	numCPUBatches = ( first_sublist_ends.size() - 1 ) * CPUFRAC;
	numGPUBatches = ( first_sublist_ends.size() - 1 ) - numCPUBatches;

    printf( "\nNumber of CPU batches: %u, Number of GPU batches: %u\n", numCPUBatches, numGPUBatches );
    assert( (numCPUBatches + numGPUBatches) == (first_sublist_ends.size() - 1) );

    // first_sublist_ends includes index 0 as first element which should be erased
    // note that this was only needed for calculation of num cpu/gpu batches.
    first_sublist_ends.erase( first_sublist_ends.begin() );
    
    // find start pivots for first sublist
    for( index = 0; index < sublist_size; index = index + BATCH_SIZE )
    {
        first_sublist_starts.push_back( index );
    }    
    
    start_vectors.push_back( first_sublist_starts );
    end_vectors.push_back( first_sublist_ends );
    
	// find remaining start and end pivot vectors for each sublist
	find_pivot_vectors( input, &start_vectors, &end_vectors, &first_sublist_ends, &list_begin_ptrs, sublist_size );

    #pragma omp parallel sections
    {
        
      // BEGIN CPU SECTION        
      #pragma omp section
      {
          // MULTIWAY MERGE ALL AT ONCE
          if( numCPUBatches > 0 )
              {
                  multiwayMerge( &input, &output_arr, numCPUBatches - 1, sublist_size, K, start_vectors, end_vectors );
              }
          tendcpu = omp_get_wtime();
      }
            
      // BEGIN GPU SECTION
      #pragma omp section
      {
          if( numGPUBatches > 0 )
              {
                  hipStream_t streams[ STREAMSPERGPU ];
                  const int NUM_THREADS_SEARCH = 4;
                  hipError_t result = hipSuccess;
                  std::vector<uint64_t> gpu_start_ptrs;
                  std::vector<uint64_t> gpu_end_ptrs;
                  uint64_t result_size = BATCH_SIZE * K * 2;
                  uint64_t stream_size = BATCH_SIZE * K * 2;
                  uint64_t *output = nullptr;
                  uint64_t *stream_dev_ptrs         = nullptr;
                  uint64_t *input_to_gpu_pinned = nullptr;
                  uint64_t *output_second = nullptr;
                  uint64_t *result_from_batches_pinned = nullptr;

                  uint64_t gpu_output_index = get_gpu_output_index( &end_vectors, numCPUBatches, NUM_THREADS_SEARCH );

                  gpu_start_ptrs.reserve( K );
                  gpu_end_ptrs.reserve( K );

                  result = create_streams( streams, STREAMSPERGPU );
                  assert( result == hipSuccess );

                  result = hipMalloc( (void**) &output, sizeof( uint64_t ) * result_size * 2 ); // 2 because we merge out of place
                  assert( result == hipSuccess );
                  output_second = output + result_size;

                  result = hipMalloc( (void**) &stream_dev_ptrs, sizeof( uint64_t ) * N );
                  assert( result == hipSuccess );

                  result = hipHostMalloc( (void**) &input_to_gpu_pinned, sizeof( uint64_t ) * BATCH_SIZE * STREAMSPERGPU );
                  assert( result == hipSuccess );

                  result = hipHostMalloc( (void**) &result_from_batches_pinned, sizeof( uint64_t ) * BATCH_SIZE * STREAMSPERGPU );
                  assert( result == hipSuccess );

                  uint64_t *output_after_rounds = K % 2 ? output_second : output;

                  for( gpu_index = numCPUBatches; gpu_index < numGPUBatches + numCPUBatches; ++gpu_index )
                      {

                          int thread_id = omp_get_thread_num();
                          int stream_id = thread_id % STREAMSPERGPU;

                          uint64_t start_index_gpu             = 0;
                          uint64_t end_index_gpu               = 0;
                          uint64_t merged_this_round           = 0;
                          uint64_t gpu_output_index_prev       = 0;
                          uint64_t gpu_output_start            = 0;
                          uint64_t gpu_output_end              = 0;

                         #pragma omp parallel for num_threads( STREAMSPERGPU ) schedule( static ) private( index, thread_id, stream_id, start_index_gpu, \
                                                                                                           end_index_gpu ) \
                             shared ( K, start_vectors, end_vectors, gpu_index, numGPUBatches, numCPUBatches, result_from_batches_pinned, \
                                      input_to_gpu_pinned, stream_dev_ptrs, output, input, gpu_output_index_prev \
                                      )                                                          \
                             reduction( +:gpu_output_index )

                          for( index = 0; index < K; index++ )
                              {

                                  uint64_t relative_index = index * sublist_size;

                                  thread_id = omp_get_thread_num();
                                  stream_id = gpu_index % STREAMSPERGPU;


                                  // copy data in BATCH_SIZE chunks from host memory to pinned memory
                                  start_index_gpu = start_vectors[ index ][ gpu_index ];
                                  end_index_gpu   = end_vectors[ index ][ gpu_index ];

                                  // calculate relative start
                                  gpu_start_ptrs[ index ] = start_index_gpu;// - relative_index;
                                  // calculate relative end index
                                  gpu_end_ptrs[ index ]   = end_index_gpu;//   - relative_index;

                                  copy_to_device_buffer( input,
                                                         input_to_gpu_pinned, stream_dev_ptrs,
                                                         streams[ stream_id ],
                                                         start_index_gpu, end_index_gpu,
                                                         BATCH_SIZE, thread_id, stream_id
                                                         );
                                  gpu_output_index += gpu_end_ptrs[ index ] - gpu_start_ptrs[ index ];
                      }
                          // do pairwise merging of sublists
                          // merge the first two sublists, after the first merge we alternate
                          // between output buffers
                          thrust::merge( thrust::device, stream_dev_ptrs + gpu_start_ptrs[ 0 ],
                              stream_dev_ptrs + gpu_end_ptrs[ 0 ],
                              stream_dev_ptrs + gpu_start_ptrs[ 1 ],
                              stream_dev_ptrs + gpu_end_ptrs[ 1 ],
                              output
                              );
                          hipDeviceSynchronize();

                          merged_this_round = gpu_end_ptrs[ 0 ] - gpu_start_ptrs[ 0 ] + \
                              gpu_end_ptrs[ 1 ] - gpu_start_ptrs[ 1 ];

                          for( index = 2; index < K; ++index )
                              {
                                  if( !( index % 2 ) )
                                      {
                                          thrust::merge( thrust::device,
                                                         output, output  + merged_this_round,
                                                         stream_dev_ptrs + gpu_start_ptrs[ index ],
                                                         stream_dev_ptrs + gpu_end_ptrs[ index ],
                                                         output_second
                                                         );
                                          hipDeviceSynchronize();
                                      }
                                  else
                                      {
                                          thrust::merge( thrust::device,
                                                         output_second, output_second + merged_this_round,
                                                         stream_dev_ptrs + gpu_start_ptrs[ index ],
                                                         stream_dev_ptrs + gpu_end_ptrs[ index ],
                                                         output
                                                         );

                                          hipDeviceSynchronize();
                                      }

                          merged_this_round += gpu_end_ptrs[ index ] - gpu_start_ptrs[ index ];
                      }

                         #pragma omp parallel for num_threads( STREAMSPERGPU ) schedule( static ) private( index, thread_id, stream_id, start_index_gpu, \
                             end_index_gpu, start_vectors, end_vectors )                         \
                             shared ( K, gpu_index, numGPUBatches, numCPUBatches, result_from_batches_pinned, \
                             input_to_gpu_pinned, stream_dev_ptrs, output_arr, input, gpu_output_index, gpu_end_ptrs, gpu_start_ptrs, \
                             gpu_output_index_prev, output_after_rounds                          \
                             )
                          for( index = 0; index < K; index++ )
                              {
                          thread_id = omp_get_thread_num();
                          stream_id = thread_id % STREAMSPERGPU;

                          // copy data in BATCH_SIZE chunks from device to host 
                          copy_from_device_buffer( output_arr + gpu_output_index_prev,
                              result_from_batches_pinned,
                              output_after_rounds,
                              streams[ stream_id ],
                              BATCH_SIZE, thread_id, stream_id,
                              &gpu_start_ptrs,
                              &gpu_end_ptrs
                              );
                      }
                          gpu_output_index_prev = gpu_output_index;
                      }

        tendgpu = omp_get_wtime();
              }
      }
    }

    // end hybrid CPU + GPU total time timer
	double tendhybrid = omp_get_wtime();

    double hybrid_total_time = tendhybrid - tstarthybrid;
    double cpu_total_time    = tendcpu    - tstartcpu;
    double gpu_total_time    = tendgpu    - tstartgpu;

    // formula given in paper
    double load_imbalance    = ( cpu_total_time - gpu_total_time ) / hybrid_total_time;

    printf( "Time CPU and GPU (total time): %f\n", hybrid_total_time );
    printf( "Time CPU Only: %f\n", cpu_total_time );
    printf( "Time GPU Only: %f\n", gpu_total_time );

    printf( "Load imbalance: %f\n", load_imbalance );

    if( !is_sorted( output_arr, N ) )
        {
            printf( "WARNING: The output array is not sorted as it should be!\n" );
        }
        

    free( input );
    free( output_arr );

	return EXIT_SUCCESS;

}
