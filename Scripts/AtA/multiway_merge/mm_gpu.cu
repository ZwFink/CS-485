#include <stdint.h>
#include <stdlib.h>
#include <stdio.h>
#include <random>
#include <algorithm> 
#include <string.h>
#include <fstream>
#include <iostream>
#include <string>
#include <math.h>
#include <queue>
#include <iomanip>
#include <set>
#include <algorithm>
#include <thread>
#include <cstdint>
#include <utility>


// thrust inclusions
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/sort.h>
#include <thrust/device_ptr.h>
#include <thrust/system/cuda/execution_policy.h> //for streams for thrust (added with Thrust v1.8)
#include <thrust/binary_search.h>
#include <thrust/execution_policy.h>


#include "mm_gpu.h"
#include "omp.h"

void warm_up_gpu( int device )
{
    hipSetDevice( device ); 		
    // initialize all ten integers of a device_vector to 1 
    thrust::device_vector<int> D(10, 1); 
    // set the first seven elements of a vector to 9 
    thrust::fill(D.begin(), D.begin() + 7, 9); 
    // initialize a host_vector with the first five elements of D 
    thrust::host_vector<int> H(D.begin(), D.begin() + 5); 
    // set the elements of H to 0, 1, 2, 3, ... 
    thrust::sequence(H.begin(), H.end()); // copy all of H back to the beginning of D 
    thrust::copy(H.begin(), H.end(), D.begin()); 
    // print D 

    printf("\nDevice: %d\n",device);

    for(int i = 0; i < D.size(); i++) 
        std::cout << " D[" << i << "] = " << D[i]; 


    // empty the vector
    D.clear();

    // deallocate any capacity which may currently be associated with vec
    D.shrink_to_fit();

    printf("\n");

    return;
}

hipError_t create_streams( hipStream_t *streams, const int num_streams )
{
    int index = 0;
    hipError_t error_code = hipSuccess;
    hipError_t result     = hipSuccess;


    for( index = 0; index < num_streams; ++index )
        {
            error_code = hipStreamCreate( &streams[ index ] );
            if(  error_code != hipSuccess )
                {
                    result = error_code;
                }
        }
    return result;
}



