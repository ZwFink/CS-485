
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <omp.h>
#include <stdlib.h>
#include <string.h>
#include <iostream>

using namespace std;

void warmUpGPU();

int main( int argc, char **argv )
{
	hipError_t error_code;
	int num_items = atoi( argv[ 1 ] );
	int upper_bound = atoi( argv[ 2 ] );

	warmUpGPU();
	const int num_trials = 3;
	int outer_index = 0;

		while( num_items < upper_bound )
		{

			for( outer_index = 0; outer_index < num_trials; outer_index++ )
			{
				char *dev_A = NULL;
				char *host_A = NULL;

				error_code = hipHostMalloc( (char **) &host_A, sizeof( char ) * num_items , hipHostMallocDefault);
				
				if( error_code != hipSuccess )
				{
					cout << "Error allocating on device" << endl;
				}

				int index = 0;
				for( index = 0; index < num_items - 1; index++ )
				{
					host_A[ index ] = 'A';
				}

				host_A[ num_items - 1 ] = '\0';

				error_code = hipMalloc( (char **) &dev_A, sizeof( char ) * num_items );

				if( error_code != hipSuccess )
				{
					cout << "Error allocating on device" << endl;
				}

				error_code = hipMemcpy( dev_A, host_A, sizeof( char ) * num_items, hipMemcpyHostToDevice );

				hipDeviceSynchronize();


				hipHostFree( host_A );
				hipFree( dev_A );
			}

			num_items += 1;

		}


	return EXIT_SUCCESS;
}

__global__ void warmup( unsigned int *tmp )
{
    if( threadIdx.x == 0 )
        {
            *tmp = 555;
        }
    return;
}

void warmUpGPU()
{
    printf( "Warming up GPU for time trialing...\n" );

    unsigned int *dev_tmp;
    unsigned int *tmp;

    hipError_t errCode = hipSuccess;


    tmp = (unsigned int *) malloc( sizeof( unsigned int ) );
    errCode = hipMalloc( (unsigned int **) &dev_tmp, sizeof( unsigned int ) );

    if( errCode != hipSuccess )
        {
            cout << "Error: dev_tmp error with code " << errCode << endl;
        }

    warmup<<<1,256>>>(dev_tmp);

    //copy data from device to host 
	errCode=hipMemcpy( tmp, dev_tmp, sizeof(unsigned int), hipMemcpyDeviceToHost);
	if(errCode != hipSuccess)
    {
        cout << "Error: getting tmp result form GPU error with code " << errCode << endl; 
	}

	hipDeviceSynchronize();

	printf("tmp (changed to 555 on GPU): %d\n",*tmp);

    hipFree(dev_tmp);

    return;

}
